#include "hip/hip_runtime.h"
bool DEBUG = true;

// Standard libraries 
#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <ctime>
#include <chrono>
#include <map>
#include <algorithm>
#include <random>
#include <cassert>
#include <memory>

// External libraries 
#include <opencv2/opencv.hpp>
#include <omp.h>
#include <hip/hip_runtime.h>
#include ""
#include <nccl.h>
#include <math.h>

// Declarations 
std::string CONFIG_PATH = "./conf/config.csv";

class Config;   
class Logger;          
class Tensor;    
class Dataset;     
class Data_loader;  
class MLP;
class Linear_layer;
class ReLU;

void load_configuration(Config& cfg);
float* allocate_array(size_t x_dim, size_t y_dim, std::string loc);

/****************************************************************************/
/******************************** Utility ***********************************/
/****************************************************************************/

struct Config {
	/* Store hyperparameters */
	size_t num_epochs = 0;
	size_t batch_size = 0;
	float lr = -1.0f;
	std::vector<size_t> hidden = {};
	std::string loc = "";
	std::string log_dir = "";
	std::string data_dir = "";
};

void load_configuration(Config& cfg) {
	/* Reads the configuration file and adds values to the configuration struct */
	std::cout << "Loading configuration...\n" << std::endl;

	// Open configuration file (CSV)
	std::ifstream cfg_file(CONFIG_PATH);

	// Read configuration file line by line.
	std::string line;
	std::string key;
	std::string value;

	while (std::getline(cfg_file, line)) {
		// Split line by comma ("key,value")
		std::istringstream ss(line);
		std::getline(ss, key, ',');
		std::getline(ss, value, ',');

		// Debugging: Print key and value (one line)
		if (DEBUG) {
			std::cout << "Key: " << key << "; Value: " << value << "\n";
		};

		// Create key-value pairs in configuration struct 
		if (key == "num_epochs") {
			cfg.num_epochs = std::stoi(value);

			// Make sure number of epochs is positive
			assert(cfg.num_epochs > 0);
		}
		else if (key == "batch_size") {
			cfg.batch_size = std::stoi(value);

			// Make sure batch size is positive
			assert(cfg.batch_size > 0);
		}
		else if (key == "lr") {
			cfg.lr = std::stof(value);

			// Make sure learning rate is positive
			assert(cfg.lr > 0.0f);
		}
		else if (key == "hidden") {
			// Value is a {x1;x2;...,xn} string which should be saved as a vector of size_t
			std::string hidden_str = value.substr(1, value.size() - 2);
			std::istringstream hidden_ss(hidden_str);
			std::string hidden_value;

			while (std::getline(hidden_ss, hidden_value, ';')) {
				cfg.hidden.push_back(std::stoi(hidden_value));

				// Make sure hidden layer sizes are positive
				assert(cfg.hidden.back() > 0);
			};
		}
		else if (key == "loc") {
			cfg.loc = value;

			// Make sure location name is either cpu, gpu or pinned
			assert(cfg.loc == "cpu" || cfg.loc == "gpu" || cfg.loc == "pinned");
		};
		else if (key == "log_dir") {
			cfg.log_dir = value;
		}
		else if (key == "data_dir") {
			cfg.data_dir = value;
		};
	};
};

/****************************************************************************/

class Logger {
	/* A logger which can save model metrics to a txt file when necessary */

public:
	// Constructor and destructor
	Logger();
	~Logger();

	// Function prototypes
	void log(int epoch, float acc);
	void log_config(Config cfg);

	// Getters
	std::string get_project_name() const { return project_name; };

private:
	// Member variables
	std::string project_name;
	std::string log_path;
	std::ofstream log_file;
};

Logger::Logger() {
	std::cout << "Initializing logger...\n" << std::endl;

	// Get current date and time in format MM-DD-HH-MM-SS (month, day, hour, min, sec) 
	auto now = std::chrono::system_clock::now();
	std::time_t now_c = std::chrono::system_clock::to_time_t(now);

	// don't use localtime because it's not thread safe
	std::tm* now_tm = std::gmtime(&now_c);

	// Convert time to string
	std::string month = std::to_string(now_tm->tm_mon + 1);
	std::string day = std::to_string(now_tm->tm_mday);
	std::string hour = std::to_string(now_tm->tm_hour);
	std::string min = std::to_string(now_tm->tm_min);
	std::string sec = std::to_string(now_tm->tm_sec);

	this->project_name = month + "-" + day + "-" + hour + "-" + min + "-" + sec;

	// Print project name
	std::cout << "Project name: " << this->project_name << "\n" << std::endl;

	// Concatenate LOG_DIR and project_name to create log path (string)
	std::string log_path = LOG_DIR + "/" + this->project_name + ".txt";

	// Open log file in append mode
	this->log_file.open(log_path, std::ios::app);
	this->log_file << "epoch,loss,accuracy,epoch_time\n";
};

Logger::~Logger() {
	/* Close log file */
	this->log_file.close();
};

void Logger::log_config(Config cfg) {
	/* Log configuration to file */
	this->log_file << "Configuration\n";
	this->log_file << "num_epochs," << cfg.num_epochs << "\n";
	this->log_file << "batch_size," << cfg.batch_size << "\n";
	this->log_file << "lr," << cfg.lr << "\n";
	this->log_file << "hidden,";

	for (int i = 0; i < cfg.hidden.size(); i++) {
		this->log_file << cfg.hidden[i] << ";";
	};

	this->log_file << "\n";
	this->log_file << "location_name," << cfg.location_name << "\n";
	this->log_file << "location_id," << cfg.location_id << "\n";
};

void Logger::log(int epoch, float acc) {
	/* Log current epoch, loss and acc */
	this->log_file << epoch << "," << acc  << "\n";
};

/****************************************************************************/

float* allocate_array(size_t x_dim, size_t y_dim, std::string loc) {
	/* Allocate either a vector or matrix as a 1D array as to favour CUDA kernel 
	   configuration (the same memory layout is used for both host and device for 
	   consistency).
	*/
	
    assert (loc == "cpu" || loc == "gpu" || loc == "pinned");

	float* array;

	// Case 1: Host (not pinned)
	if (loc == "cpu") {
		array = new float[x_dim * y_dim];
		assert (array != nullptr);
	}

	// Case 2: Host (pinned)
	else if (loc == "pinned") {
		hipHostAlloc(&array, x_dim * y_dim * sizeof(float), hipHostMallocDefault);
		assert (array != nullptr);
	}

	// Case 3: Device (not pinned)
	else if (loc == "gpu") {
		hipMalloc(&array, x_dim * y_dim * sizeof(float));
		assert (array != nullptr);
	};

	return array;
};

class Tensor {
	/* A matrix class with dynamic allocation of data on host and device */

public:
	// Constructors and destructor
	Tensor(size_t x_dim = 1, size_t y_dim = 1, std::string loc = "cpu");

	// Function prototypes
	void set_data(int mode);
	void to(std::string loc);

	// Getters 
	std::vector<size_t> get_shape() { return { x_dim, y_dim }; };
	std::string get_location() { return loc; };
	float* get_data() { return data; };

private:
	// Pointers to data (allocated on host or device heap). Note that smart pointers 
	// are not an option with CUDA (so we have to make due with regular pointers).
	// This is also why I chose wrap the data pointer in a class (Tensor) - the
	// destructor will free the memory when the object goes out of scope.
	float* data_h;
	float* data_d;

	// Meta data
	size_t x_dim, y_dim;
	std::string loc;
};

Tensor::Tensor(size_t x_dim, size_t y_dim, std::string loc) {

	this->x_dim = x_dim; 
	this->y_dim = y_dim;
	this->loc= loc;

	// Even a vector should be considered as a matrix with one row or column
	assert (x_dim > 0 && y_dim > 0);

	// Allocate matrix or vector in proper location (host or gpu or both)
	if (loc == "cpu") {
		this->data_h = allocate_array(x_dim, y_dim, "cpu");
		this->data_d = nullptr;
	}
	else if (loc == "gpu") {
		this->data_h = nullptr;
		this->data_d = allocate_array(x_dim, y_dim, "gpu");
	}
	else if (loc == "pinned") {
		this->data_h = allocate_array(x_dim, y_dim, "pinned");
		this->data_d = allocate_array(x_dim, y_dim, "gpu");
	};
};

void Tensor::set_data(int mode) {
	/* Initialize matrix as either zeros or random values (depending on input) */

	// Temporary array on host of the same size as the data array in the Tensor object
	float* new_data = new float[x_dim * y_dim];

	// If input mode is 0 (zeros), set new data to zeros
	if (mode == 0) {
		for (int i = 0; i < x_dim; i++) {
			for (int j = 0; j < y_dim; j++) {
				new_data[i * y_dim + j] = 0.0f;
			};
		};
	}

	// If input mode is 1 (random), set new data to random values between 0 and 1
	else if (mode == 1) {
		for (int i = 0; i < x_dim; i++) {
			for (int j = 0; j < y_dim; j++) {
				new_data[i * y_dim + j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
			};
		};
	};

	// If device memory is used, copy data from new data to data_d
	// and free temp memory (and remove pointer to temp data)
	if (loc == "gpu") {
		hipMemcpy(data_d, new_data, x_dim * y_dim * sizeof(float), hipMemcpyHostToDevice);
		delete new_data;
	};

	// If host memory is used, set data_h to new data and remove old pointer
	if (loc == "cpu") {
		if (data_h != nullptr) {
			delete data_h;
		};

		data_h = new_data;
	};

	// If pinned memory is used, copy new data to data_d  and make data_h point to new data
	if (loc == "pinned") {
		// Copy new data to device memory
		hipMemcpy(data_d, new_data, x_dim * y_dim * sizeof(float), hipMemcpyHostToDevice);

		// Make data_h point to new data
		if (data_h != nullptr) {
			delete data_h;
		};

		data_h = new_data;
	};
};

void Tensor::to(std::string dest) {
	/* Copy data from pinned host to device or vice versa */

	assert (this->loc == "pinned");

	if (dest == "host") {
		hipMemcpy(data_h, data_d, x_dim * y_dim * sizeof(float), hipMemcpyDeviceToHost);
	}
	else if (dest == "device") {
		hipMemcpy(data_d, data_h, x_dim * y_dim * sizeof(float), hipMemcpyHostToDevice);
	};
};

/****************************************************************************/
/******************************* Data ***************************************/
/****************************************************************************/

class Dataset {
	/* Class to store dataset information and provide methods to access data */

public:
	// Constructor 
	Dataset(std::string partition_dir);

	// Function prototypes
	std::pair<cv::Mat, int> get_item(int idx);

	// Getters
	int get_len() { return len; };
	size_t get_num_classes() { return num_classes; };
	size_t get_img_size() { return img_size; };

	// Storage on the heap (dynamic)
	std::vector<std::string> img_paths;
	std::map<std::string, int> path2label;

private:
	// Storage on the stack (fixed size)
	int len;
	size_t num_classes;
	std::string partition_dir;
	size_t img_size;
};

Dataset::Dataset(std::string partition_dir) {
	/* Find image paths and create path2label map */

	// Temporary storages 
	std::vector<std::string> all_paths;
	std::string labels_path;

	// Glob all paths in data folder (images and label file)
	cv::glob(partition_dir, all_paths, false);

	// Extract image paths and label csv path
	for (int i = 0; i < all_paths.size(); i++) {
		if (all_paths[i].find(".png") != std::string::npos) {
			this->img_paths.push_back(all_paths[i]);
		}
		if (all_paths[i].find(".csv") != std::string::npos) {
			labels_path = all_paths[i];
		};
	};

	// Set size of dataset
	this->len = img_paths.size();

	// Read labels from csv file and create path2label map
	// (ignore first line as it is a header)
	std::ifstream labels_file(labels_path);
	std::string line;

	while (std::getline(labels_file, line)) {
		if (line.find("path") == std::string::npos) {
			std::istringstream ss(line);
			std::string path;
			std::string label;

			std::getline(ss, path, ',');
			std::getline(ss, label, ',');

			// Print path and label
			this->path2label[path] = std::stoi(label);
		};
	};

	// Get unique labels
	std::vector<int> unique_labels;
	for (auto const& x : path2label) {
		unique_labels.push_back(x.second);
	};

	// Remove duplicates
	std::sort(unique_labels.begin(), unique_labels.end());
	unique_labels.erase(std::unique(unique_labels.begin(), unique_labels.end()), unique_labels.end());

	// Set number of classes
	this->num_classes = unique_labels.size();

	// Get image size (flattened)
	cv::Mat img = cv::imread(img_paths[0], cv::IMREAD_GRAYSCALE);
	this->img_size = img.rows * img.cols;
};

std::pair<cv::Mat, int> Dataset::get_item(int idx) {
	/* Get image and label at dataset (path list) index. Use unique pointer to 
	   ensure that the space on the heap is freed after it has been used for 
	   augmentation etc. in the dataloader
	*/

	std::string img_path = img_paths[idx];

	// Load image data on the heap (for cheap pass by value for wrapper class instance)
	cv::Mat img = cv::imread(img_path, cv::IMREAD_GRAYSCALE);	 // Unsigned int 8-bit (0-255)
	int label = this->path2label[img_path];

	return std::make_pair(img, label);
};

/****************************************************************************/

class Data_loader {
	/* The main function of the data loader is to provide batches of data to the model.
	   It does this by calling get_item on the dataset object to get a single image and label,
	   and then stacking these images and labels into a batch.
	*/

public:
	// Constructor
	Data_loader(Dataset dataset, size_t batch_size, bool shuffle, std::string loc);

	// Function prototypes
	void prepare_epoch();
	auto get_batch(int batch_idx);
	size_t get_num_batches() { return this->batch_indices.size(); };

private:
	// Storage on the stack
	size_t batch_size;
	size_t num_classes;
	bool shuffle;
	Dataset dataset;

	// Storge on the heap 
	std::vector<int> obs_indices;
	std::vector<std::vector<int>> all_batch_indices;
	std::vector<int> batch_indices;
	size_t img_size;

	// Tensors for images (flattened) and labels (one-hot encoded)
	Tensor X;
	Tensor y;
};

Data_loader::Data_loader(Dataset dataset, size_t batch_size, bool shuffle, std::string loc) : dataset{ dataset }, batch_size{ batch_size }, shuffle{ shuffle } {
	std::cout << "Initializing data loader...\n" << std::endl;

	this->batch_size = batch_size;
	this->shuffle = shuffle;
	this->dataset = dataset;

	// Get image size (flattened) and number of classes from dataset
	this->img_size = dataset.get_img_size();
	this->num_classes = dataset.get_num_classes();

	// Create indices (list of integers) from 0 to len(dataset)
	this->obs_indices.reserve(dataset.get_len());

	for (int i = 0; i < dataset.get_len(); i++) {
		this->obs_indices.push_back(i);  
	};

	// Create batches of indices (list of lists)
	prepare_epoch();

	// Allocate pinned memory for images and labels on host (standard and last batch) and if 
	// main location is device, then keep same size memory allocated (so new batches
	// only update the storage when moved from cpu to device - no reallocation). 
	X = Tensor(this->img_size, this->batch_size, "pinned");
	y = Tensor(this->num_classes, this->batch_size, "pinned");
};

void Data_loader::prepare_epoch() {
	/* Shuffle indices and split into batches */

	// Shuffle indices
	std::random_device rnd_device;
	std::mt19937 mersenne_engine{ rnd_device() }; 
	std::shuffle(begin(this->obs_indices), end(this->obs_indices), mersenne_engine);

	// Split into batches of size batch_size (last batch might be smaller)
	for (int i = 0; i < this->obs_indices.size(); i += this->batch_size) {
		std::vector<int> batch(this->obs_indices.begin() + i, this->obs_indices.begin() + i + this->batch_size);
		this->all_batch_indices.push_back(batch);
	};

	assert (this->all_batch_indices[0].size() == this->batch_size);
};

auto Data_loader::get_batch(int batch_idx) {
	// Get observation indices for batch
	this->batch_indices = this->all_batch_indices[batch_idx];
 
	// Update tensors with new batch data (flat images, labels)  
	// [NUMA optimized due to where the function is first called]
	for (int obs_idx = 0; obs_idx < this->batch_size; obs_idx++) { 
		// Get image and label from dataset 
		std::pair<cv::Mat, int> obs = this->dataset.get_item(batch_indices[obs_idx]);
		cv::Mat img = obs.first;
		int label = obs.second;

		// Convert image to float [0;1] 
		img.convertTo(img, CV_32F, 1.0 / 255.0);

		// One-hot encode label and save as y tensor data 
		for (int i = 0; i < this->num_classes; i++) {
			y.get_data()[i][obs_idx] = (i == label) ? 1.0f : 0.0f;
		};
		
		// Flatten image and and save as X tensor data 
		for (int i = 0; i < img.rows; i++) {
			for (int j = 0; j < img.cols; j++) {
				X.get_data()[i * img.cols + j][obs_idx] = img.at<float>(i, j);
			};
		};
	};

	// Return pair of tensors (images and labels) 
	return std::make_pair(X, y); 
};

/****************************************************************************/
/******************************** Model *************************************/
/****************************************************************************/

class ReLU {
	/* Class to store activation layer information and perform forward and backward passes */

public:
	// Constructor
	ReLU(size_t in_features, std::string loc, size_t batch_size);

	// Function prototypes
	Tensor forward(Tensor in);
	Tensor backward(Tensor grad);

private:
	// Hyperparameters and meta data
	std::string loc;
	size_t in_features;
	size_t batch_size;

	// Kernel configurations
	dim3 grid_size_fw;
	dim3 block_size_fw;

	dim3 grid_size_bw;
	dim3 block_size_bw;

	// Tensor storages
	Tensor in;
	Tensor out;
	Tensor grad_in;
};

ReLU::ReLU(size_t in_features, std::string loc, size_t batch_size) : in_features{ in_features }, loc{ loc }, batch_size{ batch_size } {
	std::cout << "Initializing ReLU layer...\n" << std::endl;

	// Allocate input, output and gradient tensors
	this->in = Tensor(in_features, batch_size, loc);
	this->out = Tensor(in_features, batch_size, loc);
	this->grad_in = Tensor(in_features, batch_size, loc);

	// Initialize grad to zeros (important)
	this->grad_in.set_data(0);

	// Initialize input and output to zeros (for debugging purposes)
	this->in.set_data(0);
	this->out.set_data(0);
};

void ReLU::reset_grads() {
	/* Reset gradients to zeros */
	grad_in.set_data(0);
};

Tensor ReLU::forward(Tensor x) {
	/* Perform forward pass on batch */

	// Save input for backward pass
	this->in = x;

	// Perform forward pass
	if (in.get_location() == "cpu") {
		_relu_host_fw(in.get_data(), out.get_data(), in_features, batch_size);
	}
	else if (in.get_location() == "gpu") {
		_relu_device_fw<<<this->grid_size_fw, this->block_size_bw>>>(in.get_data(), out.get_data(), in_features, batch_size);
	};

	return out;
}

Tensor ReLU::backward(Tensor grad_out) {
	/* Perform backward pass on batch */

	if (grad_out.get_location() == "host") {
		_relu_host_bw(in.get_data(), out.get_data(), grad_out.get_data(), grad_in.get_data(), in_features, batch_size);
	}
	else if (grad.get_location().get_name() == "device") {
		_relu_device_bw<<<this->grid_size_bw, this->block_size_bw>>>(in.get_data(), out.get_data(), grad_out.get_data(), grad_in.get_data(), in_features, batch_size);
	};

	return grad_in;
};

__host__ void _relu_host_fw(float* t_in, float* t_out, size_t in_features, size_t batch_size) {
	/* Perform forward pass on batch (ReLU) */

	for (int i = 0; i < in_features; i++) {
		for (int j = 0; j < batch_size; j++) {
			t_out[i * batch_size + j] = (t_in[i * batch_size + j] > 0) ? t_in[i * batch_size + j] : 0;
		};
	};
};

__global__ void _relu_host_bw(float* t_in, float* t_out, float* t_grad_in, float* t_grad_out, size_t in_features, size_t batch_size) {
	/* Perform backward pass on batch (ReLU) */

	for (int i = 0; i < in_features; i++) {
		for (int j = 0; j < batch_size; j++) {
			t_grad_in[i * batch_size + j] = (t_out[i * batch_size + j] > 0) ? t_grad_out[i * batch_size + j] : 0;
		};
	};
};

__host__ void _relu_device_fw(float* t_in, float* t_out, size_t in_features, size_t batch_size) {
	

};

__global__ void _relu_device_bw(float* t_in, float* t_out, float* t_grad_in, float* t_grad_out, size_t in_features, size_t batch_size) {

};

/****************************************************************************/

class Linear_layer {
	/* Class to store layer information and perform forward and backward passes */

public:
	// Constructor
	Linear_layer(size_t in_features, size_t out_features, std::string loc, size_t batch_size);

	// Function prototypes
	Tensor forward(Tensor in);
	Tensor backward(Tensor grad, Tensor grad_weights, Tensor grad_bias);

	// Getters
	std::vector<Tensor> get_weights { return weights; };
	std::vector<Tensor> get_bias { return bias; };

	std::vector<Tensor> get_grad_weights { return grad_weights; };
	std::vector<Tensor> get_grad_bias { return grad_bias; };

private:
	// Hyperparameters and meta data
	size_t in_features, out_features;
	size_t max_batch_size;
	std::string loc;
	bool pinned_output;

	// Kernel configurations
	dim3 grid_size_fw1;
	dim3 block_size_fw1;

	dim3 grid_size_fw2;
	dim3 block_size_fw2;

	dim3 grid_size_bw1;
	dim3 block_size_bw1;

	dim3 grid_size_bw2;
	dim3 block_size_bw2;

	// Tensor storages
	Tensor weights;   // W.T
	Tensor bias;      // b

	Tensor in;	      // x
	Tensor out1;      // W.T * x 
	Tensor out2;      // W.T * x + b

	Tensor grad_weight;  // dW
	Tensor grad_bias;     // db

	Tensor grad_in;      // dx
	Tensor grad_out1;      // d(W.T * x)
};

Linear_layer::Linear_layer(size_t in_features, size_t out_features, std::string loc, size_t batch_size) : loc{ loc }, in_features{ in_features }, out_features{ out_features }, max_batch_size{ batch_size } {
	std::cout << "Initializing layer...\n" << std::endl;

	assert (in_features > 0 && out_features > 0 && batch_size > 0);
	assert (loc == "cpu" || loc == "gpu");

	// Allocate weights and biases 
	this->weights = Tensor(out_features, in_features, loc);
	this->bias = Tensor(out_features, 1, loc);

	// Allocate output tensor 
	this->out1 = Tensor(out_features, batch_size, loc);

	if (loc == "cpu") {
		this->out2 = Tensor(out_features, batch_size, "cpu");
		this->pinned_output = false;
	}
	else if (loc == "gpu") {
		this->out2 = Tensor(out_features, batch_size, "pinned");
		this->pinned_output = true;
	};

	// Allocate gradient of loss w.r.t. parameters tensors 
	this->grad_weight = Tensor(out_features, in_features, loc);
	this->grad_bias = Tensor(out_features, 1, loc);

	// Allocate gradient of loss w.r.t. input tensor (step 1 and 2)
	this->grad_in = Tensor(in_features, batch_size, loc);
	this->grad_out1 = Tensor(out_features, batch_size, loc);  // TODO: Is this correct size?

	// Initialize weights, bias and gradients to zeros (important)
	this->weights.set_data(1);
	this->bias.set_data(0);

	this->grad_weight.set_data(0);
	this->grad_bias.set_data(0);

	this->grad_in.set_data(0);
	this->grad_out1.set_data(0);

	// Initialize input and step outputs as zeros (only for debugging purposes)
	this->in.set_data(0);

	this->out1.set_data(0);
	this->out2.set_data(0);

	// Compute optimal kernel configuration for forward pass (step 1)



	// Compute optimal kernel configuration for forward pass (step 2)


	// Compute optimal kernel configuration for backward pass (step 1)


	// Compute optimal kernel configuration for backward pass (step 2)


};

void Linear_layer::reset_grads() {
	/* Reset gradients to zeros */

	grad_weight.set_data(0);
	grad_bias.set_data(0);
	grad_in.set_data(0);
	grad_out1.set_data(0);
};

Tensor Linear_layer::forward(Tensor x) {
	/* Perform forward pass on batch where step 1 is out1=(W.T * x) and step 2 is out2=(out1 + expand(b)) */

	// Save x for backward pass
	this->in = x;

	// Perform forward pass 
	if (in.get_location() == "cpu") {
		_ll_host_fwd_1(in.get_data(), weights.get_data(), out1.get_data(), in_features, out_features, batch_size);
		_ll_host_fwd_2(out1.get_data(), bias.get_data(), out2.get_data(), in_features, out_features, batch_size);
	}

	else if (in.get_location() == "gpu") {
		_ll_device_fwd_1<<<this->grid_size_fw1, this->block_size_fw1>>>(in.get_data(), weights.get_data(), out1.get_data(), in_features, out_features, batch_size);
		_ll_device_fwd_2<<<this->grid_size_fw2, this->block_size_fw2>>>(out1.get_data(), bias.get_data(), out2.get_data(), in_features, out_features, batch_size);
	};

	return out2;
};

Tensor Linear_layer::backward(Tensor grad_out) {
	/* Perform backward pass */

	if (grad.get_location()== "cpu") {
		_ll_host_bwd_2(out1.get_data(), bias.get_data(), out2.get_data(), grad_bias.get_data(), grad_out2.get_data(), grad_out1.get_data(), in_features, out_features, batch_size);
		_ll_host_bwd_1(in.get_data(), weights.get_data(), out1.get_data(), grad_weights.get_data(), grad_out1.get_data(), grad_in.get_data(), in_features, out_features, batch_size);
	}

	else if (grad.get_location() == "gpu") {
		_ll_device_bwd_2<<<this->grid_size_bw2, this->block_size_bw2>>>(out1.get_data(), bias.get_data(), out2.get_data(), grad_bias.get_data(), grad_out2.get_data(), grad_out1.get_data(), in_features, out_features, batch_size);
		_ll_device_bwd_1<<<this->grid_size_bw1, this->block_size_bw1>>>(in.get_data(), weights.get_data(), out1.get_data(), grad_weights.get_data(), grad_out1.get_data(), grad_in.get_data(), in_features, out_features, batch_size);
	};

	return grad_in;
};

__host__ void _ll_host_fwd(float* t_in, float* t_weights, float* t_out1, size_t in_features, size_t out_features, size_t batch_size) {
	/* Perform forward pass on batch (all steps of linear layer) */

	// W.T * x
	for (int i = 0; i < out_features; i++) {
		for (int j = 0; j < batch_size; j++) {
			t_out1[i * batch_size + j] = 0.0f;

			for (int k = 0; k < in_features; k++) {
				t_out1[i * batch_size + j] += t_weights[i * in_features + k] * t_in[k * batch_size + j];
			};
		};
	};

	// (W.T * x) + b 
	for (int i = 0; i < out_features; i++) {
		for (int j = 0; j < batch_size; j++) {
			t_out1[i * batch_size + j] += t_bias[i];
		};
	};
};

__host__ void _ll_host_bwd(float* t_out1, float* t_bias, float* t_out2, float* t_grad_bias, float* t_grad_out1, size_t in_features, size_t out_features, size_t batch_size) {
	/* Perform backward pass on batch (all steps of linear layer) */

	// Compute gradient of loss w.r.t. bias
	for (int i = 0; i < out_features; i++) {
		t_grad_bias[i] = 0.0f;

		for (int j = 0; j < batch_size; j++) {
			t_grad_bias[i] += t_grad_out1[i * batch_size + j];
		};
	};

	// Compute gradient of loss w.r.t. weights
	for (int i = 0; i < out_features; i++) {
		for (int j = 0; j < in_features; j++) {
			t_grad_weights[i * in_features + j] = 0.0f;

			for (int k = 0; k < batch_size; k++) {
				t_grad_weights[i * in_features + j] += t_grad_out1[i * batch_size + k] * t_in[j * batch_size + k];
			};
		};
	};

	// Compute gradient of loss w.r.t. input
	for (int i = 0; i < in_features; i++) {
		for (int j = 0; j < batch_size; j++) {
			t_grad_in[i * batch_size + j] = 0.0f;

			for (int k = 0; k < out_features; k++) {
				t_grad_in[i * batch_size + j] += t_weights[k * in_features + i] * t_grad_out1[k * batch_size + j];
			};
		};
	};


};

__global__ void _ll_device_fwd_1(float* t_in, float* t_weights, float* t_out1, size_t in_features, size_t out_features, size_t batch_size) {
	
};

__global__ void _ll_device_fwd_2(float* t_out1, float* t_bias, float* t_out2, size_t in_features, size_t out_features, size_t batch_size) {
	
};

__global__ void _ll_device_bwd_1(float* t_in, float* t_weights, float* t_out1, float* t_grad_weights, float* t_grad_out1, size_t in_features, size_t out_features, size_t batch_size) {
	
};

__global__ void _ll_device_bwd_2(float* t_out1, float* t_bias, float* t_out2, float* t_grad_bias, float* t_grad_out1, size_t in_features, size_t out_features, size_t batch_size) {
	
};

/****************************************************************************/

class MLP {
	/* Class to store neural network configuration and perform forward and backward passes */

public:
	// Constructor
	MLP(size_t in_dim, std::vector<size_t> hidden_dims, size_t out_dim, std::string loc, size_t batch_size);

	// Function prototypes
	Tensor forward(Tensor x);
	Tensor backward(Tensor grad);

	// Getters
	std::vector<Tensor> get_weights() { return layer_weights; };
	std::vector<Tensor> get_biases() { return layer_bias; };

	std::vector<Tensor> get_grad_weights() { return layer_grad_weights; };
	std::vector<Tensor> get_grad_biases() { return layer_grad_bias; };

private:
	// Hyperparameters and meta data
	std::string loc;
	size_t batch_size;

	size_t in_dim;
	std::vector<size_t> hidden_dims;
	size_t out_dim;

	// Module lists 
	std::vector<Linear_layer> layers;
	std::vector<ReLU> activations;

	// Tensor storages
	std::vector<Tensor> layer_weights;
	std::vector<Tensor> layer_bias;

	std::vector<Tensor> layer_grad_weights;
	std::vector<Tensor> layer_grad_bias;

	Tensor out;
	Tensor grad_out;
};

MLP::MLP(size_t in_dim, std::vector<size_t> hidden_dims, size_t out_dim, std::string loc, size_t batch_size) : in_dim{ in_dim }, hidden_dims{ hidden_dims }, out_dim{ out_dim }, loc{ loc }, batch_size{ batch_size } {
	std::cout << "Initializing neural network...\n" << std::endl;

	// Initialize network layers and activations
	layers.push_back(Linear_layer(in_dim, hidden_dims[0], loc, batch_size));
	activations.push_back(ReLU(hidden_dims[0], loc, batch_size));

	for (int i = 0; i < hidden_dims.size() - 1; i++) {
		layers.push_back(Linear_layer(hidden_dims[i], hidden_dims[i + 1], loc, batch_size));
		activations.push_back(ReLU(hidden_dims[i+1], loc, batch_size));
	};

	layers.push_back(Linear_layer(hidden_dims.back(), out_dim, loc, batch_size));

	// Get weight tensors for each layer and collect in vector
	for (int i = 0; i < layers.size(); i++) {
		layer_weights.push_back(layers[i].get_weights());
	}

	// Get bias tensors for each layer and collect in vector
	for (int i = 0; i < layers.size(); i++) {
		layer_bias.push_back(layers[i].get_bias());
	}

	// Get gradient of weights tensors for each layer and collect in vector
	for (int i = 0; i < layers.size(); i++) {
		layer_grad_weights.push_back(layers[i].get_grad_weights());
	}

	// Get gradient of bias tensors for each layer and collect in vector
	for (int i = 0; i < layers.size(); i++) {
		layer_grad_bias.push_back(layers[i].get_grad_bias());
	}
};

void MLP::reset_grads() {
	/* Reset gradients to zeros */

	for (int i = 0; i < layers.size() - 1; i++) {
		layers[i].reset_grads();
		activations[i].reset_grads();
	};

	layers.back().reset_grads();
};

Tensor MLP::forward(Tensor x) {
	/* Perform forward pass */

	out = x;

	for (int i = 0; i < layers.size() - 1; i++) {
		out = layers[i].forward(out);
		out = activations[i].forward(out);
	};

	out = layers.back().forward(out);

	return out;
};

Tensor MLP::backward(Tensor grad_out) {
	/* Perform backward pass */

	for (int i = layers.size() - 1; i >= 1; i--) {
		this->grad_out = activations[i].backward(this->grad_out);
		this->grad_out = layers[i].backward(this->grad_out);
	};

	this->grad_out = layers[0].backward(this->grad_out);

	return this->grad_out;
};

/****************************************************************************/
/******************************* Loss ***************************************/
/****************************************************************************/

class Cross_entropy {
	/* Class to store loss function information and perform forward and backward passes */

public:
	// Constructor
	Cross_entropy(size_t in_features, std::string loc, size_t batch_size);

	// Function prototypes
	Tensor forward(Tensor x, Tensor y);
	Tensor backward(Tensor grad_out);

private:
	// Hyperparameters and meta data
	size_t num_classes;
	size_t batch_size;
	std::string loc;

	// Kernel configurations
	dim3 grid_size_fw1;
	dim3 block_size_fw1;

	dim3 grid_size_fw2;
	dim3 block_size_fw2;

	dim3 grid_size_fw3;
	dim3 block_size_fw3;

	dim3 grid_size_fw4;
	dim3 block_size_fw4;

	dim3 grid_size_fw5;
	dim3 block_size_fw5;

	// Tensor storages
	Tensor pred_logits;     // Logits from model
	Tensor true_probs;      // One-hot encoded labels

	Tensor out1;            
	Tensor out2;
	Tensor out3;
	Tensor out4;
	Tensor out5;            // Final output (loss)

	Tensor grad_out1;
	Tensor grad_out2;
	Tensor grad_out3;
	Tensor grad_out4;
	Tensor grad_out5;       // Gradient of loss w.r.t. logits (input)
};

Cross_entropy::Cross_entropy(size_t num_classes, std::string loc, size_t batch_size) : loc { loc }, num_classes{ num_classes }, batch_size{ batch_size } {
	std::cout << "Initializing Cross-entropy loss function...\n" << std::endl;

	// Allocate intermediate output tensors (not pinned - only used for computation)
	this->out1 = Tensor(num_classes, batch_size, loc);
	this->out2 = Tensor(num_classes, batch_size, loc);
	this->out3 = Tensor(num_classes, batch_size, loc);
	this->out4 = Tensor(num_classes, batch_size, loc);

	// Allocate final output (loss) tensor (pinned if device is main location)
	if (loc == "cpu") {
		this->out5 = Tensor(1, batch_size, "cpu");
	}
	else if (loc == "gpu") {
		this->out5 = Tensor(1, batch_size, "pinned");
	};

	// Allocate gradient tensors (not pinned - only used for computation)
	this->grad_out1 = Tensor(num_classes, batch_size, loc);
	this->grad_out2 = Tensor(num_classes, batch_size, loc);
	this->grad_out3 = Tensor(num_classes, batch_size, loc);
	this->grad_out4 = Tensor(num_classes, batch_size, loc);
	this->grad_out5 = Tensor(num_classes, batch_size, loc);

	// Initialize grad to zeros (important)
	this->grad_out1.set_data(0);
	this->grad_out2.set_data(0);
	this->grad_out3.set_data(0);
	this->grad_out4.set_data(0);
	this->grad_out5.set_data(0);

	// Initialize output to zeros (for debugging purposes)
	this->out1.set_data(0);
	this->out2.set_data(0);
	this->out3.set_data(0);
	this->out4.set_data(0);
	this->out5.set_data(0);

	// Compute optimal kernel configuration for forward pass (step 1)

	// Compute optimal kernel configuration for forward pass (step 2)

	// Compute optimal kernel configuration for forward pass (step 3)

	// Compute optimal kernel configuration for forward pass (step 4)

	// Compute optimal kernel configuration for forward pass (step 5)

};

void Cross_entropy::reset_grads() {
	/* Reset gradients to zeros */

	this->grad_out1.set_data(0);
	this->grad_out2.set_data(0);
	this->grad_out3.set_data(0);
	this->grad_out4.set_data(0);
	this->grad_out5.set_data(0);
};

Tensor Cross_entropy::forward(Tensor x, Tensor y) {
	/* Perform forward pass through a computationally stable version of cross 
	   entropy operating on logits instead of softmax output (using log-sum-exp 
	   trick). 

	   General cross entropy: L = -sum(y_j * log(yhat_j))

	   The output from the model is not softmaxed, but rather the logits. The
	   softmax function is numerically unstable for large values, so we use the
	   log-sum-exp trick to compute the softmax probabilities.

	   Numerically unstable softmax: log(yhat_j) = log(exp(x_j) / sum(exp(x_k)))
	   Numerically stable softmax:   log(yhat_j) = x_j - log(sum(exp(x_k)))

	   Note that here the term (log(sum(exp(x_k))) is the same for all classes
	   (used later). Insert log(yhat_j) into the cross entropy formula:

	   Stable cross entropy: L = -sum(y_j * (x_j - log(sum(exp(x_k))))

	   To compute this loss using CUDA efficiently we need to split the forward
	   pass into multiple kernels (steps) similar to how we used 2 steps in the
	   implementation of the linear layer. For each prediction (one-hot-encoding)
	   in the batch we need to compute the loss using the following steps:

	   Step 1: Compute the exp(x) vector (using element-wise kernel) 
	   Step 2: Compute the s_1 = sum(exp(x_k)) scalar (using reduction kernel)
	   Step 3: Compute the s_2 = log(s) scalar (apply log to scalar from step 2)
	   Step 4: Compute y_j * (x_j - s_2) (using element-wise/dot kernel)
	   Step 5: Compute the sum of the loss (using reduction kernel again)

	   The forward kernel configuration can be chosen so that the number of threads
	   is equal to the number of classes (num_classes) and the number of blocks is
	   equal to the number of observations in the batch (batch_size). This way we
	   can compute the loss for each observation in the batch in parallel.
	*/

	// Save logits and labels for backward pass
	this->pred_logits = x;
	this->true_probs = y;

	// Forward pass
	if (x.get_location() == "cpu") {
		_ce_host_fwd_1(pred_logits.get_data(), out1.get_data(), num_classes, batch_size);
		_ce_host_fwd_2(out1.get_data(), out2.get_data(), num_classes, batch_size);
		_ce_host_fwd_3(out2.get_data(), out3.get_data(), num_classes, batch_size);
		_ce_host_fwd_4(out3.get_data(), out4.get_data(), true_probs.get_data(), num_classes, batch_size);
		_ce_host_fwd_2(out4.get_data(), out5.get_data(), num_classes, batch_size);
	}
	else if (x.get_location() == "gpu") {
		_ce_device_fwd_1<<<grid_size_fwd_1, block_size_fwd_1>>>(pred_logits.get_data(), out1.get_data(), num_classes, batch_size);
		_ce_device_fwd_2<<<grid_size_fwd_2, block_size_fwd_2>>>(out1.get_data(), out2.get_data(), num_classes, batch_size);
		_ce_device_fwd_3<<<grid_size_fwd_3, block_size_fwd_3>>>(out2.get_data(), out3.get_data(), num_classes, batch_size);
		_ce_device_fwd_4<<<grid_size_fwd_4, block_size_fwd_4>>>(out3.get_data(), out4.get_data(), true_probs.get_data(), num_classes, batch_size);
		_ce_device_fwd_2<<<grid_size_fwd_5, block_size_fwd_5>>>(out4.get_data(), out5.get_data(), num_classes, batch_size);
	};

	return out5;
};

Tensor Cross_entropy::backward(Tensor grad_out5) {
	/* Perform backward pass through computationally stable cross entropy loss. 
	   To match the forward pass, the backward pass is split into 5 steps:

	   Step 5: Compute the gradient of the loss w.r.t. the logits (x_j - s_2)
	   Step 4: Compute the gradient of the loss w.r.t. the scalar s_2
	   Step 3: Compute the gradient of the loss w.r.t. the sum of exp(x_k)
	   Step 2: Compute the gradient of the loss w.r.t. the exp(x_k) vector
	   Step 1: Compute the gradient of the loss w.r.t. the logits (x_j)
	*/
	if (grad.get_location().get_name() == "cpu") {
		_ce_host_bwd_5(out4.get_data(), grad_out5.get_data(), num_classes, batch_size);
		_ce_host_bwd_4(out3.get_data(), out4.get_data(), grad_out4.get_data(), num_classes, batch_size);
		_ce_host_bwd_3(out2.get_data(), out3.get_data(), grad_out3.get_data(), num_classes, batch_size);
		_ce_host_bwd_2(out1.get_data(), out2.get_data(), grad_out2.get_data(), num_classes, batch_size);
		_ce_host_bwd_1(pred_logits.get_data(), out1.get_data(), grad_out1.get_data(), num_classes, batch_size);	
	}
	else if (grad.get_location().get_name() == "gpu") {
		_ce_device_bwd_5<<<grid_size_bwd_5, block_size_bwd_5>>>(out4.get_data(), grad_out5.get_data(), num_classes, batch_size);
		_ce_device_bwd_4<<<grid_size_bwd_4, block_size_bwd_4>>>(out3.get_data(), out4.get_data(), grad_out4.get_data(), num_classes, batch_size);
		_ce_device_bwd_3<<<grid_size_bwd_3, block_size_bwd_3>>>(out2.get_data(), out3.get_data(), grad_out3.get_data(), num_classes, batch_size);
		_ce_device_bwd_2<<<grid_size_bwd_2, block_size_bwd_2>>>(out1.get_data(), out2.get_data(), grad_out2.get_data(), num_classes, batch_size);
		_ce_device_bwd_1<<<grid_size_bwd_1, block_size_bwd_1>>>(pred_logits.get_data(), out1.get_data(), grad_out1.get_data(), num_classes, batch_size);
	};

	return grad_out1;
};

__host__ float* _ce_host_fwd(float* t_out4, float* t_out5, size_t num_classes, size_t batch_size) {
	/* Compute all steps of the forward pass of the stable cross entropy loss: L = -sum_j(y_j * (x_j - log(sum_k(exp(x_k))))*/

	// Compute the sum of exp(x_k) for each observation in the batch
	for (int i = 0; i < batch_size; i++) {
		float sum_exp = 0.0f;
		for (int j = 0; j < num_classes; j++) {
			sum_exp += exp(t_out4[i * num_classes + j]);
		}
		t_out5[i] = sum_exp;
	}

	// Compute the loss for each observation in the batch
	for (int i = 0; i < batch_size; i++) {
		float loss = 0.0f;
		for (int j = 0; j < num_classes; j++) {
			float y_j = t_out1[i * num_classes + j];
			float x_j = t_pred_logits[i * num_classes + j];
			float log_sum_exp = log(t_out5[i]);
			loss -= y_j * (x_j - log_sum_exp);
		}
		t_out4[i] = loss;
	}

	return t_out4;








};

__host__ float* _ce_host_bwd(float* t_pred_logits, float* t_out1, float* t_grad_out1, size_t num_classes, size_t batch_size) {
	

};

__global__ float* _ce_device_fwd_1(float* t_pred_logits, float* t_out1, size_t num_classes, size_t batch_size) {
	/* Step 1: Compute the exp(x) vector (using element-wise kernel) */

};

__global__ float* _ce_device_fwd_2(float* t_out1, float* t_out2, size_t num_classes, size_t batch_size) {
	/* Step 2: Compute the s_1 = sum(exp(x_k)) scalar (using reduction kernel) */

};

__global__ float* _ce_device_fwd_3(float* t_out2, float* t_out3, size_t num_classes, size_t batch_size) {
	/* Step 3: Compute the s_2 = log(s) scalar (apply log to scalar from step 2) */

};

__global__ float* _ce_device_fwd_4(float* t_out3, float* t_out4, float* t_true_probs, size_t num_classes, size_t batch_size) {
	/* Step 4: Compute y_j * (x_j - s_2) (using element-wise/dot kernel) */

};

__global__ float* _ce_device_bwd_1(float* t_pred_logits, float* t_out1, float* t_grad_out1, size_t num_classes, size_t batch_size) {
	/* Step 1: Compute the gradient of the loss w.r.t. the logits (x_j) */

};

__global__ float* _ce_device_bwd_2(float* t_out1, float* t_out2, float* t_grad_out2, size_t num_classes, size_t batch_size) {
	/* Step 2: Compute the gradient of the loss w.r.t. the exp(x_k) vector */

};

__global__ float* _ce_device_bwd_3(float* t_out2, float* t_out3, float* t_grad_out3, size_t num_classes, size_t batch_size) {
	/* Step 3: Compute the gradient of the loss w.r.t. the sum of exp(x_k) */

};

__global__ float* _ce_device_bwd_4(float* t_out3, float* t_out4, float* t_grad_out4, size_t num_classes, size_t batch_size) {
	/* Step 4: Compute the gradient of the loss w.r.t. the scalar s_2 */

};

/****************************************************************************/
/******************************* Optimizer **********************************/
/****************************************************************************/

class SGD {
	/* Class to store optimizer information and perform optimization steps */

public:
	// Constructor
	SGD(float lr, MLP* model, Cross_entropy* loss_fn, std::string loc);

	// Function prototypes
	void step();
	void reset_grads();

private:
	// Store  hyperparameters and meta data
	float lr;
	std::string loc;

	// Store reference to model and loss function
	MLP* model;
	Cross_entropy* loss_fn;

	// Kernel configurations

	// Tensor storages
	std::vector<Tensor> weight;
	std::vector<Tensor> bias;
	std::vector<Tensor> grad_weight;
	std::vector<Tensor> grad_bias;
};

SGD::SGD(float lr, MLP* model, Cross_entropy* loss_fn, std::string loc) : lr{ lr }, loc{ loc } model{ model }, loss_fn{ loss_fn } {
	std::cout << "Initializing optimizer...\n" << std::endl;

	// Get weights and biases from model
	weights = model->get_weights();
	bias = model->get_biases();

	// Get gradients of weights and biases from model
	grad_weights = model->get_grad_weights();
	grad_bias = model->get_grad_biases();
};

void SGD::reset_grads() {
	/* Reset gradients to zero */

	this->model->reset_grads();
	this->loss_fn->reset_grads();
};

void SGD::step() {
	/* Perform optimization step (after backward has been called) */

	if (loc.get_name() == "cpu") {
		for (int i = 0; i < weights.size(); i++) {
			_update_host(weights[i], grad_weights[i], bias[i], grad_bias[i], lr);
		};
	}
	else if (loc.get_name() == "gpu") {
		for (int i = 0; i < weights.size(); i++) {
			_weight_update_device<<<grid_size, block_size_weights>>>(weights[i], grad_weights[i], lr);
			_bias_update_device<<<grid_size, block_size_bias>>>(bias[i], grad_bias[i], lr);
		};
	};
};

__host__ void _update_host(float* t_weights, float* t_grad_weights, float* t_bias, float* t_grad_bias, float lr) {
	/* Update weights and biases on host */

};

__global__ void update_device(float* t_weights, float* t_grad_weights, float lr) {
	/* Update weights on device */

};


/****************************************************************************/
/****************************************************************************/

void process(Config cfg, Data_loader data_loader, MLP model, Cross_entropy loss_fn, \
	         SGD optimizer, Logger logger, std::string mode) {
	/* Train or test the neural network */
	std::cout << "Training neural network...\n" << std::endl;

	// Setup variables
	int epoch_size = data_loader.get_num_batches();

	// Storage 
	float epoch_acc = 0;

	for (int batch_idx = 0; batch_idx < epoch_size; batch_idx++) {
		// Get batch: Pointers to host memory locations (pinned)
		auto batch = data_loader.get_batch(batch_idx);    
		Tensor imgs = batch.first;
		Tensor labels = batch.second;

		// TODO: hipMemcpyAsync (above) is non-blocking so we need to synchronize here
		//       but to use it we need to make sure that the data is not being used in
		//       the forward pass (which is the case here). Thus, we would need to have
		// 	     a separate buffer for the data that we can copy to and from (so that we
		//       can copy the data to the buffer and then copy from the buffer to the
		//       pinned memory). This would require a separate buffer for each batch
		//       which is not optimal. We could also use streams to overlap computation
		//       and data transfer but this is more complex and requires more memory
		//       as we need to have multiple streams for each batch.


		// Move from pinned host to device memory
		if (main_location.get_name() == "device") {
			x.copy_to_device();
			y.copy_to_device();
		};

		// Forward pass
		auto logits = model.forward(imgs);
		auto loss = loss_fn.forward(logits, labels);

		// Backward pass and optimization step 
		if (mode == "train") {
			model.backward(loss_fn.backward(1.0f));
			optimizer.step();
			optimizer.reset_grads();
		};

		// Move from device to pinned host memory
		if (main_location.get_name() == "device") {
			logits.copy_to_host();
			labels.copy_to_host();
		};

		// Batch accuracy and loss
		float batch_acc = accuracy(logits, labels);
		
		hipDeviceSynchronize();

		epoch_acc += batch_acc.to("cpu").get_data()[0] 
	};

	// Average epoch loss and accuracy
	epoch_acc /= epoch_size;

	// Log epoch loss and accuracy
	int idx = (mode == "train") ? epoch_idx + 1 : -1;
	logger.log(idx, epoch_acc);
};

/****************************************************************************/

int main() {
	double t_begin, t_end;
	t_begin = omp_get_wtime();

	// Logger (for epoch metrics)
	Logger logger;

	// Hyperparameters 
	Config cfg;
	load_configuration(cfg);
	logger.log_config(cfg);

	// Dataset partitions (train and test)
	std::string train_dir = DATA_DIR + "/train";
	std::string test_dir = DATA_DIR + "/test";

	Dataset train_set(train_dir);
	Dataset test_set(test_dir);

	// Initialize Loaders for data partitions. Note that we already know allocate 
	// the host memory for the batches so we don't have to reallocate every new batch.
	// Note also that this does not mess with later NUMA optimization as we do not
	// have first touch before .get_item(idx) is called (where we copy the data from
	// the dataset). If main location is device, we keep the same size memory allocated
	// (so new batches only update the storage when moved from cpu to device - no reallocation)
	Data_loader train_loader(train_set, cfg.batch_size, true, cfg.loc);
	Data_loader test_loader(test_set, cfg.batch_size, false, cfg.loc);

	// Neural network (always initialize on host). CPU parallelization of weight/bias 
	// initialization. This will not be NUMA optimized as all threads needs access to
	// all weights and biases (so we do not benefit from spreading them out with CPU
	// parallelized first touch). I don't initialize on device as PyTorch also does not
	// do that.
	MLP model(train_set.get_img_size(), cfg.hidden, train_set.get_num_classes(), cfg.loc, cfg.batch_size);

	// Initialize loss function. Note that we allocate backward pass data tensors for the 
	// loss in main_location as we don't initialize this data (no learnable params in cross 
	// entropy loss) unlike the model which have weights that are initialized to specfic values.
	Cross_entropy loss_fn(cfg.hidden.back(), cfg.loc, cfg.batch_size);

	// Initialize optimizer. We initialize it with pointers to the model parameters 
	// in the main location so that we can always access them when we do updates with .step()
	SGD optimizer(cfg.lr, model.get_weights(), model.get_biases(), model.get_grad_weights(), model.get_grad_biases(), cfg.loc);

	// Print setup time
	t_end = omp_get_wtime();
	std::cout << "Setup time: " << (t_end - t_begin) << " seconds\n";

	// Training phase
	process(cfg, train_loader, model, loss_fn, optimizer, logger, "train");

	double avg_time = 0;
	for (int epoch_idx = 1; epoch_idx < cfg.num_epochs; epoch_idx++) {  
		t_begin = omp_get_wtime();
		process(cfg, train_loader, model, loss_fn, optimizer, logger, "train");
		t_end = omp_get_wtime();

		avg_time += t_end - t_begin;
	};

	avg_time /= (cfg.num_epochs - 1);

	std::cout << "Average training epoch time: " << avg_time << " seconds\n";

	// Testing phase
	t_begin = omp_get_wtime();
	process(cfg, train_loader, model, loss_fn, optimizer, logger, "test");
	t_end = omp_get_wtime();
	std::cout << "Total testing time: " << (t_end - t_begin) << " seconds\n";

	return 0;
};




// https://luniak.io/cuda-neural-network-implementation-part-1/ (fwd/bwd parallel)
// https://stackoverflow.com/questions/53498952/tensorflow-horovod-nccl-and-mpi (nccl and MPI data parallism)
// https://github.com/olcf-tutorials/MPI_ping_pong (cuda-aware MPI)


